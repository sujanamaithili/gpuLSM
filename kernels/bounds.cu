#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <bounds.cuh>

template <typename Key, typename Value>
__device__ int lowerBound(int level, Key key, int bufferSize, Pair<Key, Value>* m) {
    int offset = 0;
    int level_size = bufferSize << level;

    for (int i = 0; i < level; ++i) {
        offset += bufferSize << i;  
    }

    Pair<Key, Value>* levelData =  m + offset;

    int left = 0;
    int right = level_size;

    while (left < right) {
        int mid = left + (right - left) / 2;
        if (levelData[mid].first < key) {
            left = mid + 1;
        } else {
            right = mid;
        }
    }

    return left;
}

template <typename Key, typename Value>
__device__ int upperBound(int level, Key key, int bufferSize, Pair<Key, Value>* m) {
    int offset = 0;
    int level_size = bufferSize << level;

    for (int i = 0; i < level; ++i) {
        offset += bufferSize << i;  
    }

    Pair<Key, Value>* levelData =  m + offset;

    int left = 0;
    int right = level_size;

    while (left < right) {
        int mid = left + (right - left) / 2;
        if (levelData[mid].first <= key) {
            left = mid + 1;
        } else {
            right = mid;
        }
    }

    return left;
}




template <typename Key>
__global__ void findBounds(int* d_l, int* d_u, const Key* k1, const Key* k2, int* d_init_count, int bufferSize, Pair<Key, Value>* m, int numLevels) {
    int queryId = blockIdx.x;
    int level = threadIdx.x;

    Key key1 = k1[queryId];
    Key key2 = k2[queryId];

    d_l[queryId * numLevels + level] = lowerBound(level, key1, bufferSize, m);
    d_u[queryId * numLevels + level] = upperBound(level, key2, bufferSize, m);

    d_init_count[queryId * numLevels + level] = d_u[queryId * numLevels + level] - d_l[queryId * numLevels + level];

}

template __global__ void findBounds<int, int>( int*, int*, const Key*, const Key*, int*, int);

