#include "lsm.cuh"
#include "query.cuh"
#include "merge.cuh"
#include <cstdio>
#include <hip/hip_runtime.h>

template <typename Key, typename Value>
__host__ __device__ lsmTree<Key, Value>::lsmTree(int numLevels, int bufferSize) {
    this->numLevels = numLevels;
    this->bufferSize = bufferSize;
    this->maxSize = 0;

    // Calculate maxSize based on the number of levels and buffer size.
    for (int i = 0; i < numLevels; ++i) {
        this->maxSize += (bufferSize << i);
    }

    // Allocate memory on the GPU.
    hipError_t status = hipMalloc(&memory, maxSize * sizeof(Pair<Key, Value>));
    if (status != hipSuccess) {
        printf("Error allocating memory for LSM tree: %s\n", hipGetErrorString(status));
        #ifndef __CUDA_ARCH__
        // Only call exit if on the host
        exit(1);
        #endif
    }
}

template <typename Key, typename Value>
__host__ __device__ lsmTree<Key, Value>::~lsmTree() {
    if (memory != nullptr) {
        hipFree(memory);
    }
}

template <typename Key, typename Value>
__host__ bool lsmTree<Key, Value>::updateKeys(const Pair<Key, Value>* kv, int batch_size)
{
    
    Pair<Key, Value>* d_buffer;
    hipMalloc(&d_buffer, batch_size * sizeof(Pair<Key, Value>));
    hipMemcpy(d_buffer, kv, batch_size * sizeof(Pair<Key, Value>), hipMemcpyHostToDevice);
    hipMalloc(&tempd_buffer, batch_size * sizeof(Pair<Key, Value>));
    hipcub::DeviceRadixSort::SortPairs(tempd_buffer, batch_size, d_buffer, d_buffer, batch_size);

    int offset = 0;
    int level_size = batch_size; //b
    int current_level = 0;
    int merged_size = batch_size;
    
    Pair<Key, Value>* m = getMemory();

    while(getNumBatches() & (1 << currentLevel)){
        Pair<Key, Value>* cur = getMemory() + offset;

        merged_size += level_size;

        d_buffer = merge(m + offset, level_size, d_buffer, level_size);
        hipMemset(cur, 0, level_size * sizeof(Pair<Key, Value>));

        offset += level_size;
        current_level++;
        level_size <<= 1
    }
    
    hipMemcpy(m + offset, d_buffer, merged_size * sizeof(Pair<Key, Value>), hipMemcpyDeviceToDevice);
    incrementBatchCounter();
    hipFree(d_buffer);
    return true;
}


template <typename Key, typename Value>
__host__ void lsmTree<Key, Value>::queryKeys(const Key* keys, int size, Value* results, bool* foundFlags) {
    // Allocate device memory for keys, results, and found flags
    Key* d_keys;
    Value* d_results;
    bool* d_foundFlags;

    hipMalloc(&d_keys, size * sizeof(Key));
    hipMalloc(&d_results, size * sizeof(Value));
    hipMalloc(&d_foundFlags, size * sizeof(bool));
    hipMemcpy(d_keys, keys, size * sizeof(Key), hipMemcpyHostToDevice);

    // Get device pointer to the LSM tree memory
    Pair<Key, Value>* d_memory = getMemory();
    int num_levels = numLevels;
    int buffer_size = bufferSize;

    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    queryKeysKernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_keys, d_results, d_foundFlags, size, d_memory, num_levels, buffer_size);

    // Copy results back to host
    hipMemcpy(results, d_results, size * sizeof(Value), hipMemcpyDeviceToHost);
    hipMemcpy(foundFlags, d_foundFlags, size * sizeof(bool), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_keys);
    hipFree(d_results);
    hipFree(d_foundFlags);
}
